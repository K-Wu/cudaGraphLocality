#include "hip/hip_runtime.h"
﻿#include "stdio.h"
#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <vector>
#include "helper_timer.h"
#include "hip/hip_runtime_api.h"
#include "helper_string.h"
#if MY_CUDA_ARCH_IDENTIFIER >= 800 // assuming 3090
#define N 687865856
#define NUM_CASCADING 8
#define NUM_PARTITION 2048 // each arry occupies 1.28125MB
#define GRIDDIM 82
#else
#define N 50000000 //assuming 2070max-q
#define NUM_CASCADING 10
#define NUM_PARTITION 150 // tuned such that kernel takes a few microseconds
#define GRIDDIM 36
#endif


#include <cstdlib>
#include <cmath>

template <int NCASCADING, int NLEN>
int verify_saxpy(float* output, float* input){
	float timer = pow(1.23, NCASCADING);
	int result = 0;
	for (size_t idx=0;idx<NLEN;idx++){
		if(abs(output[idx]-input[idx]*timer)>0.000002){
			printf("Error: %lu %f %f!\n",idx, output[idx], input[idx]*timer);
			result++;
		}
	}
	return result;
}

void random_initialize(float* arr, size_t len)
{
	for (size_t idx = 0; idx < len; idx++)
	{
		arr[idx] = (std::rand() + 0.0) / RAND_MAX;
	}
	return;
}

inline void __checkCudaErrors(hipError_t err, const char* file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		//exit(-1);

	}
	//return err;
}
#define checkCudaErrors(err) (__checkCudaErrors((err),__FILE__,__LINE__))

template <int NPARTITION, int NLEN>
__global__ void shortKernel(float* vector_d, float* in_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	for (int curr_idx = idx; curr_idx < NLEN / NPARTITION; curr_idx += blockDim.x * gridDim.x) {
		__stcg(&vector_d[curr_idx], 1.23 * __ldlu(&in_d[curr_idx]));
	}
}

template <int NPARTITION, int NCASCADING, int NLEN>
__global__ void shortKernel_merged(float* vectors_d[NCASCADING+1], int ipartition) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x + NLEN/NPARTITION * ipartition;
	for (int i_cascading = 0; i_cascading < NCASCADING; i_cascading++) {
		for (int curr_idx = idx; curr_idx < NLEN / NPARTITION * (ipartition+1); curr_idx += blockDim.x * gridDim.x) {
			vectors_d[i_cascading+1][curr_idx] = 1.23 * vectors_d[i_cascading][curr_idx];
		}
	}
}

template <int NPARTITION, int NCASCADING, int NLEN>
__global__ void shortKernel_merged_optimized(float* vectors_d[NCASCADING+1], int ipartition) {
	long long idx = blockIdx.x * blockDim.x + threadIdx.x + NLEN/NPARTITION * ipartition;
	for (int i_cascading = 0; i_cascading < NCASCADING; i_cascading++) {
		for (int curr_idx = idx; curr_idx < NLEN / NPARTITION * (ipartition+1); curr_idx += blockDim.x * gridDim.x) {
			__stcg(&vectors_d[i_cascading+1][curr_idx], 1.23 * __ldlu(&vectors_d[i_cascading][curr_idx]));
		}
	}
}

struct param_resetStreamAccessPolicyWindow {
	struct hipAccessPolicyWindow accessPolicyWindow;
	hipStream_t stream;
};

void resetStreamAccessPolicyWindow(void* param) {
	struct hipAccessPolicyWindow accessPolicyWindow = ((struct param_resetStreamAccessPolicyWindow*)param)->accessPolicyWindow;
	hipStream_t stream = ((struct param_resetStreamAccessPolicyWindow*)param)->stream;

	hipLaunchAttributeValue attr;
	attr.accessPolicyWindow.base_ptr = accessPolicyWindow.base_ptr;
	attr.accessPolicyWindow.num_bytes = accessPolicyWindow.num_bytes;
	// hitRatio causes the hardware to select the memory window to designate as persistent in the area set-aside in L2
	attr.accessPolicyWindow.hitRatio = accessPolicyWindow.hitRatio;
	// Type of access property on cache hit
	attr.accessPolicyWindow.hitProp = hipAccessPropertyNormal;
	// Type of access property on cache miss
	attr.accessPolicyWindow.missProp = hipAccessPropertyNormal;
	checkCudaErrors(hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &attr));
}

template <int NPARTITION, int NCASCADING, bool FLAG_ENABLE_L2_POLICY>
int __main_01() {
	hipGraph_t graph;
	hipGraphExec_t instance;
	hipStream_t stream;
	hipKernelNodeParams kernelNodeParams;
	hipGraphNode_t kernel_node[NCASCADING];
	hipGraphNode_t host_nodes[NCASCADING - 1];
	float* input;
	input = (float*) malloc(sizeof(float) * N);
	random_initialize(input, N);
	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	checkCudaErrors(hipGraphCreate(&graph, 0));
	float* vector_d[NCASCADING + 1];
	for (int idx = 0; idx < NCASCADING + 1; idx++) {
		checkCudaErrors(hipMalloc(&vector_d[idx], sizeof(float) * N));
	}
	checkCudaErrors(hipMemcpy(vector_d[0], input,sizeof(float)*N, hipMemcpyHostToDevice));
	checkCudaErrors(hipStreamSynchronize(stream));
	StopWatchInterface* timerExec = NULL;
	sdkCreateTimer(&timerExec);
	sdkStartTimer(&timerExec);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
//first iteration of ipartition: create graph then execute
	for (int iCascade = 0; iCascade < NCASCADING; iCascade++) {
		std::vector<hipGraphNode_t> node_dependencies;
		if (iCascade != 0) {
#if MY_CUDA_ARCH_IDENTIFIER >= 800
#ifdef HOST_NODE_CANNOT_EXECUTE_CUDA_FUNCTION
			if constexpr (FLAG_ENABLE_L2_POLICY) {
				hipHostNodeParams hostNodeParams;
				hostNodeParams.fn = resetStreamAccessPolicyWindow;
				struct param_resetStreamAccessPolicyWindow host_params;
				hipKernelNodeAttrValue last_kernel_node_attribute;
				checkCudaErrors(hipGraphKernelNodeGetAttribute(kernel_node[iCascade - 1], hipKernelNodeAttributeAccessPolicyWindow, &last_kernel_node_attribute));
				host_params.accessPolicyWindow = last_kernel_node_attribute.accessPolicyWindow;
				host_params.stream = stream;
				hostNodeParams.userData = (void*)&host_params;
				std::vector<hipGraphNode_t> host_node_dependencies = { kernel_node[iCascade - 1] };
				checkCudaErrors(hipGraphAddHostNode(&host_nodes[iCascade - 1], graph, host_node_dependencies.data(), host_node_dependencies.size(), &hostNodeParams));
				node_dependencies.push_back(host_nodes[iCascade - 1]);
			}
			else {
#endif
				node_dependencies.push_back(kernel_node[iCascade - 1]);
#ifdef HOST_NODE_CANNOT_EXECUTE_CUDA_FUNCTION
			}
#endif
#else
			node_dependencies.push_back(kernel_node[iCascade - 1]);
#endif
		}
		void* kernelArgsPtr[2] = { (void*)&vector_d[iCascade+1],(void*)&vector_d[iCascade] };
		kernelNodeParams.func = (void*)shortKernel<NPARTITION, N>;
		kernelNodeParams.gridDim = GRIDDIM;
		kernelNodeParams.blockDim = 1024;
		kernelNodeParams.kernelParams = (void**)&kernelArgsPtr;
		kernelNodeParams.extra = NULL;
		kernelNodeParams.sharedMemBytes = 0;
		checkCudaErrors(hipGraphAddKernelNode(&kernel_node[iCascade], graph, node_dependencies.data(), node_dependencies.size(), &kernelNodeParams));
		
	}
	checkCudaErrors(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
	checkCudaErrors(hipGraphLaunch(instance, stream));
	checkCudaErrors(hipStreamSynchronize(stream));
	for (int ipartition = 1; ipartition < NPARTITION; ipartition++) {
		for (int iCascade = 0; iCascade < NCASCADING; iCascade++) {
			//replace parameter
			hipKernelNodeParams kernelNodeParams_curr;
			float* kernelArgs_curr[2] = { &vector_d[iCascade + 1][N / NPARTITION * ipartition],&vector_d[iCascade][N / NPARTITION * ipartition] };
			void* kernelArgsPtr_curr[2] = { (void*)&kernelArgs_curr[0], (void*)&kernelArgs_curr[1] };
			kernelNodeParams_curr.func = (void*)shortKernel<NPARTITION, N>;
			kernelNodeParams_curr.gridDim = GRIDDIM;
			kernelNodeParams_curr.blockDim = 1024;
			kernelNodeParams_curr.kernelParams = (void**)&kernelArgsPtr_curr;
			kernelNodeParams_curr.extra = NULL;
			kernelNodeParams_curr.sharedMemBytes = 0;
#if MY_CUDA_ARCH_IDENTIFIER >= 800
			if constexpr(FLAG_ENABLE_L2_POLICY){
				hipKernelNodeAttrValue node_attribute;                                     // Kernel level attributes data structure
				node_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void*>(&vector_d[iCascade + 1][N / NPARTITION * ipartition]); // Global Memory data pointer
				node_attribute.accessPolicyWindow.num_bytes = N / NPARTITION * sizeof(float);                    // Number of bytes for persistence access.
																							// (Must be less than hipDeviceProp_t::accessPolicyMaxWindowSize)
				node_attribute.accessPolicyWindow.hitRatio = 0.6;                          // Hint for cache hit ratio
				node_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting; // Type of access property on cache hit
				node_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;  // Type of access property on cache miss.

				//Set the attributes to a CUDA Graph Kernel node of type hipGraphNode_t
				checkCudaErrors(hipGraphKernelNodeSetAttribute(kernel_node[iCascade], hipKernelNodeAttributeAccessPolicyWindow, &node_attribute));
				checkCudaErrors(hipGraphExecKernelNodeSetParams(instance, kernel_node[iCascade], &kernelNodeParams_curr));
#ifdef HOST_NODE_CANNOT_EXECUTE_CUDA_FUNCTION
				//TODO: set graph host node attribute
				if (iCascade != NCASCADING) {
					struct param_resetStreamAccessPolicyWindow params_host_curr;
					params_host_curr.stream = stream;
					params_host_curr.accessPolicyWindow = node_attribute.accessPolicyWindow;
					hipHostNodeParams hostNodeParams;
					hostNodeParams.fn = resetStreamAccessPolicyWindow;
					hostNodeParams.userData = (void*)&params_host_curr;
					hipGraphHostNodeSetParams(host_nodes[iCascade], &hostNodeParams);
				}
#endif
			}
			else {
				checkCudaErrors(hipGraphExecKernelNodeSetParams(instance, kernel_node[iCascade], &kernelNodeParams_curr));
			}
#else
			checkCudaErrors(hipGraphExecKernelNodeSetParams(instance, kernel_node[iCascade], &kernelNodeParams_curr));
#endif
		}
		checkCudaErrors(hipGraphLaunch(instance, stream));
		checkCudaErrors(hipStreamSynchronize(stream));
	}
	hipEventRecord(stop);

	sdkStopTimer(&timerExec);
	printf("Execution time: %f (ms)\n", sdkGetTimerValue(&timerExec));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Execution time (GPU): %f (ms)\n", milliseconds);
	checkCudaErrors(hipGraphExecDestroy(instance));
	checkCudaErrors(hipGraphDestroy(graph));
	checkCudaErrors(hipStreamDestroy(stream));
	float* output = (float*) malloc(sizeof(float)*N);
	checkCudaErrors(hipMemcpy(output, vector_d[NCASCADING],sizeof(float)*N,hipMemcpyDeviceToHost));
	printf("Errors: %d\n",verify_saxpy<NCASCADING,N>(output, input));
	return 0;
}

int main0() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortKernel<NUM_PARTITION), N>, hipFuncCachePreferShared);
	return __main_01<NUM_PARTITION, NUM_CASCADING, false>();
}

int main1() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortKernel<1), N>, hipFuncCachePreferShared);
	return __main_01<1, NUM_CASCADING, false>();
}

int main3() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortKernel<NUM_PARTITION), N>, hipFuncCachePreferShared);
	return __main_01<NUM_PARTITION, NUM_CASCADING, true>();
}

int main4() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortKernel<1), N>, hipFuncCachePreferShared);
	return __main_01<1, NUM_CASCADING, true>();
}

template <int NPARTITION, int NCASCADING, bool FLAG_OPTIMIZATION, bool FLAG_BASELINE>
int __main2() {
	float* input;
	input = (float*)malloc(sizeof(float) * N);
	random_initialize(input, N);
	float* vectors_d[NCASCADING + 1];
	float** vectors_d_d;
	for (int idx = 0; idx < NCASCADING + 1; idx++) {
		checkCudaErrors(hipMalloc(&vectors_d[idx], sizeof(float) * N));
	}
	checkCudaErrors(hipMalloc(&vectors_d_d, sizeof(float*)*(NCASCADING+1)));
	checkCudaErrors(hipMemcpy(vectors_d[0], input, sizeof(float) * N, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(vectors_d_d, vectors_d, sizeof(float*) * (NCASCADING + 1), hipMemcpyHostToDevice));
	checkCudaErrors(hipStreamSynchronize(0));
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	StopWatchInterface* timerExec = NULL;
	sdkCreateTimer(&timerExec);
	sdkStartTimer(&timerExec);
	hipEventRecord(start);
	for (int ipartition = 0; ipartition < NPARTITION; ipartition++) {
		if constexpr(FLAG_BASELINE){
			for (int i_cascading=0;i_cascading<NCASCADING;i_cascading++){
				shortKernel<NPARTITION, N><<<GRIDDIM,1024>>>(&vectors_d[i_cascading+1][N/NPARTITION*ipartition],&vectors_d[i_cascading+0][N/NPARTITION*ipartition]);
			}
		}
		else{
			if constexpr(FLAG_OPTIMIZATION){
				shortKernel_merged_optimized<NPARTITION, NUM_CASCADING, N><<<GRIDDIM,1024>>>(vectors_d_d, ipartition);
			}
			else{
				shortKernel_merged<NPARTITION, NUM_CASCADING, N><<<GRIDDIM,1024>>>(vectors_d_d, ipartition);
			}
		}
	}
	hipEventRecord(stop);
	checkCudaErrors(hipStreamSynchronize(0));
	sdkStopTimer(&timerExec);
	printf("Execution time: %f (ms)\n", sdkGetTimerValue(&timerExec));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Execution time (GPU): %f (ms)\n", milliseconds);
	float* output = (float*) malloc(sizeof(float)*N);
	checkCudaErrors(hipMemcpy(output, vectors_d[NCASCADING],sizeof(float)*N,hipMemcpyDeviceToHost));
	printf("Errors: %d\n",verify_saxpy<NCASCADING,N>(output, input));
	return 0;
}

int main2() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortKernel_merged<NUM_PARTITION), NUM_CASCADING, N>, hipFuncCachePreferShared);
	return __main2<NUM_PARTITION, NUM_CASCADING, false, false>();
}

int main5() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortKernel_merged<NUM_PARTITION), NUM_CASCADING, N>, hipFuncCachePreferShared);
	return __main2<NUM_PARTITION, NUM_CASCADING, true, false>();
}

int main6() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortKernel_merged<NUM_PARTITION), NUM_CASCADING, N>, hipFuncCachePreferShared);
	return __main2<NUM_PARTITION, NUM_CASCADING, false, true>();
}

int main7() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(shortKernel_merged<1), NUM_CASCADING, N>, hipFuncCachePreferShared);
	return __main2<1, NUM_CASCADING, false, true>();
}

int main(int argc, char** argv) {
	#if MY_CUDA_ARCH_IDENTIFIER >= 800 
		printf("cuda arch >= 800\n");
	#endif
	if (checkCmdLineFlag(argc, (const char**)argv, "help")) {
		printf("Command line: jacobiCudaGraphs [-option]\n");
		printf("Valid options:\n");
		printf(
			"-gpumethod=<0,1 or 2>  : 0 - [Default] CUDA Graph Paritioned");
		printf("                       : 1 - CUDA Graph\n");
		printf("                       : 2 - Non CUDA Graph\n");
	}
	int gpumethod = -1;
	if (checkCmdLineFlag(argc, (const char**)argv, "gpumethod")) {
		gpumethod = getCmdLineArgumentInt(argc, (const char**)argv, "gpumethod");
		if (gpumethod < 0 || gpumethod > 7) {
			printf("Error: gpumethod must be 0 or 1 or 2 or 3 or 4, gpumethod = %d is invalid\n", gpumethod);
			exit(EXIT_SUCCESS);
		}

	}
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);
	switch (gpumethod) {
	case 0:
		main0();
		break;
	case 1:
		main1();
		break;
	case 2:
		main2();
		break;
	case 3:
		main3();
		break;
	case 4:
		main4();
		break;
	case 5:
		main5();
		break;
	case 6:
		main6();
		break;
	case 7:
		main7();
		break;
	}
	sdkStopTimer(&timer);
	printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
}